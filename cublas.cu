#include "hip/hip_runtime.h"
#include "helper.h"
#include <iostream>
#include "hipblas.h"

void cublas_multiplication(const double* A, double* C, const int rows, const int columns) {
	Timer t;
	/*
	 * Cublass uses column-major implementation contrary to c/c++ default row-major
	 * For reference about mn,k,lda,ldb,ldc,alpha,beta official reference here:
	 * http://www.netlib.org/blas/dgemm.f
	 */
	int m = columns, n = columns, k = rows;
	int lda = rows, ldb = rows, ldc = columns;
	// C = alpha*A*B + beta*C
	const double alpha = 1, beta = 0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasStatus_t status;
	t.start_count();

	status=hipblasDgemm(handle,
		HIPBLAS_OP_T,  // A**T
		HIPBLAS_OP_N,  // A 
		m, n, k, &alpha,
		A, lda,
		A, ldb,
		&beta, C, ldc);
	t.stop_count();
	if(status!=HIPBLAS_STATUS_SUCCESS)
	{
		std::cout << "Error in cublas operation";
		exit(EXIT_FAILURE);
	}
	std::cout << "Time elapsed to multiply using hipblasDgemm is " << t.time()<<" ms"<< std::endl<<std::endl;
	hipblasDestroy(handle);
}
