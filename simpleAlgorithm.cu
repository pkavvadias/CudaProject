#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Multiplications.cuh"

__global__ void simple_algorithm(const double* A, double* C, const int rows, const int columns) {

	double element = 0.0;
	const int 	row = blockIdx.y * blockDim.y + threadIdx.y,
	            col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < rows && col < columns)
	{
		for (int k = 0; k < rows; ++k)
		{
			element += A[k * columns + row] * A[k * columns + col];
		}
		C[row * columns + col] = element;
	}

}